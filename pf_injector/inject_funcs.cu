#include "hip/hip_runtime.h"
/*
 * Copyright 2020, NVIDIA CORPORATION.
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 * 
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */


#include <cstdint>
#include <cstdio>

#include "nvbit_reg_rw.h"
#include "utils/utils.h"
#include "pf_injector.h"
#include "arch.h"

extern "C" __device__ __noinline__ void
inject_error(uint64_t piinfo, uint64_t pverbose_device, int destGPRNum, int regval, int numDestGPRs, int maxRegs) {
    /**
     * EDIT FOR FLEX GRIP INJECTION
     * I only need to identify the fault site
     * "instruction", "LANEID", "warp_id", "SMID"
     * The good thing is that I can control most of the things from the host side
     * That is, the instruction and the instrumentation is for each instruction
     */
    auto inj_info = (inj_info_t *) piinfo;
    uint32_t verbose_device = *((uint32_t *) pverbose_device);
    auto sm_id = get_smid();
    if (sm_id != inj_info->injSMID)
        return; // This is not the selected SM. No need to proceed.

    auto lane_id = get_laneid();
    if (lane_id != inj_info->injLaneID)
        return; // This is not the selected Lane ID. No need to proceed.

    auto warp_id = get_warpid();
    if (warp_id != inj_info->warpID)
        return; // This is not the selected Warp ID
    /**********************************************************************/

    assert(numDestGPRs > 0);
    uint32_t injAfterVal = 0;
    uint32_t injBeforeVal = nvbit_read_reg(destGPRNum); // read the register value
    if (DUMMY) {
        injAfterVal = injBeforeVal;
    } else {
//        injAfterVal = injBeforeVal ^ inj_info->injMask;
        nvbit_write_reg(destGPRNum, int32_t(inj_info->injMask));
    }
    // updating counter/flag to check whether the error was injected
    if (verbose_device)
        printf("register=%d, before=0x%x, after=0x%x, expected_after=0x%x\n", destGPRNum, injBeforeVal,
               nvbit_read_reg(destGPRNum), injAfterVal);
    inj_info->errorInjected = true;
    atomicAdd((unsigned long long *) &inj_info->injNumActivations, 1LL);
}
