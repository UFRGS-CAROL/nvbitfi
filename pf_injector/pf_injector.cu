#include "hip/hip_runtime.h"
/*
 * Copyright 2020, NVIDIA CORPORATION.
 * 
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 * 
 *     http://www.apache.org/licenses/LICENSE-2.0
 * 
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cstdint>
#include <cstdio>
#include <cassert>
#include <pthread.h>
#include <string>
#include <fstream>
#include <vector>
#include <algorithm>
#include <iostream>
#include <csignal>
#include <unordered_set>

#include "nvbit_tool.h"
#include "nvbit.h"
#include "utils/utils.h"

#include "globals.h"
#include "pf_injector.h"

/**
 * New functions and variables for FLexGrip Injection
 */
#define FATAL(error) throw std::runtime_error(std::string("ERROR ") + __FILE__ + ":" + std::to_string(__LINE__));

int verbose;
__managed__ int verbose_device;
int limit = INT_MAX;

// injection parameters input filename: This file is created the script
// that launched error injections (MUST BE GLOBAL)
std::string injInputFilename;

pthread_mutex_t mutex;

//__managed__ inj_info_t inj_info;

__managed__ inj_info_t *managed_inj_info_array = nullptr;

/* Set used to avoid re-instrumenting the same functions multiple times */
std::unordered_set<hipFunction_t> already_instrumented;


void reset_inj_info(int i = 0) {
    managed_inj_info_array[i].injInstType = 0;
    managed_inj_info_array[i].injSMID = 0;
    managed_inj_info_array[i].injLaneID = 0;
    managed_inj_info_array[i].injMask = 0;
    managed_inj_info_array[i].injNumActivations = 0;
    managed_inj_info_array[i].errorInjected = false;
    managed_inj_info_array[i].warpID = 0;
}

// for debugging 
void print_inj_info(int i = 0) {
    assert(fout.good());
    std::cout << "InstType=" << managed_inj_info_array[i].injInstType << ", SMID=" << managed_inj_info_array[i].injSMID
              << ", LaneID=" << managed_inj_info_array[i].injLaneID
              << ", WarpID=" << managed_inj_info_array[i].warpID;
    std::cout << ", Mask=" << managed_inj_info_array[i].injMask << std::endl;
}

// Parse error injection site info from a file. This should be done on host side.
void parse_params(const std::string &filename, int i = 0) {
    static bool parse_flag = false; // file will be parsed only once - performance enhancement
    if (!parse_flag) {
        parse_flag = true;
        reset_inj_info();

        std::ifstream ifs(filename, std::ifstream::in);
        if (ifs.is_open()) {
            ifs >> managed_inj_info_array[i].injSMID;
            assert(managed_inj_info_array[i].injSMID < 1000); // we don't have a 1000 SM system yet.

            ifs >> managed_inj_info_array[i].injLaneID;
            assert(managed_inj_info_array[i].injLaneID < 32); // Warp-size is 32 or less today.

            ifs >> managed_inj_info_array[i].injMask;

            ifs >> managed_inj_info_array[i].injInstType; // instruction type
            assert(managed_inj_info_array[i].injInstType <=
                   NUM_ISA_INSTRUCTIONS); // ensure that the value is in the expected range

        } else {
            printf(" File %s does not exist!", filename.c_str());
            printf(" This file should contain enough information about the fault site to perform "
                   "a permanent error injection run: ");
            printf("(1) SM ID, (2) Lane ID (within a warp), (3) 32-bit mask (as int32), "
                   "(4) Instruction type (as integer, see maxwell_pascal.h). \n");
            assert(false);
        }
        ifs.close();

        if (verbose) {
            print_inj_info();
        }
    }
}

// Parse error injection site info from a file. This should be done on host side.
void parse_flex_grip_file(const std::string &filename) {
    std::ifstream input_file(filename);
    std::vector<inj_info_t> host_database_inj_info;
    if (input_file.good()) {
        // Read the file that contains the error model from FlexGrip
        while (!input_file.eof()) {
            std::string line, word;
            std::vector<std::string> row;
            // read an entire row and
            // store it in a string variable 'line'
            std::getline(input_file, line);
            if (input_file.eof()) {
                break;
            }
            // used for breaking words
            std::stringstream s(line);

            // read every column data of a row and
            // store it in a string variable, 'word'
            while (std::getline(s, word, ';')) {
                // add all the column data
                // of a row to a vector
                row.push_back(word);
            }
            //add to the vector the faulty values
            inj_info_t new_inj_info;
            new_inj_info.injInstType = std::stoul(row[0]);
            new_inj_info.injLaneID = std::stoul(row[1]);
            new_inj_info.warpID = std::stoul(row[2]);
            new_inj_info.injSMID = std::stoul(row[3]);
            auto faulty_out = std::stoul(row[4]);
            auto golden_out = std::stoul(row[5]);
            new_inj_info.injMask = faulty_out ^ golden_out;
            host_database_inj_info.push_back(new_inj_info);
        }
        // COPY to gpu the array of injections
        CUDA_SAFECALL(hipMallocManaged(&managed_inj_info_array,
                                        host_database_inj_info.size() * sizeof(inj_info_t)));
        std::copy(host_database_inj_info.begin(), host_database_inj_info.end(), managed_inj_info_array);
        CUDA_SAFECALL(hipDeviceSynchronize());
    } else {
        FATAL("Not possible to open the file " + filename)
    }
}

void update_verbose() {
    static bool update_flag = false; // update it only once - performance enhancement
    if (!update_flag) {
        update_flag = true;
        hipDeviceSynchronize();
        verbose_device = verbose;
        hipDeviceSynchronize();
    }
}

int get_maxregs(hipFunction_t func) {
    int maxregs = -1;
    hipFuncGetAttribute(&maxregs, HIP_FUNC_ATTRIBUTE_NUM_REGS, func);
    return maxregs;
}

void INThandler(int sig) {
    signal(sig, SIG_IGN); // disable Ctrl-C

    fout << ":::NVBit-inject-error; ERROR FAIL Detected Singal SIGKILL;";
    fout << " injNumActivations: " << managed_inj_info_array[0].injNumActivations << ":::";
    fout.flush();
    exit(-1);
}


/* nvbit_at_init() is executed as soon as the nvbit tool is loaded. We typically
 * do initializations in this call. In this case for instance we get some
 * environment variables values which we use as input arguments to the tool */
// DO NOT USE UVM (__managed__) variables in this function
void nvbit_at_init() {
    /* just make sure all managed variables are allocated on GPU */
    setenv("CUDA_MANAGED_FORCE_DEVICE_ALLOC", "1", 1);
    injInputFilename = "nvbitfi-injection-info.txt";
    /* we get some environment variables that are going to be use to selectively
     * instrument (within a interval of kernel indexes and instructions). By
     * default we instrument everything. */
    if (getenv("TOOL_VERBOSE")) {
        verbose = std::stoi(getenv("TOOL_VERBOSE"));
    } else {
        verbose = 0;
    }

    if (getenv("INPUT_INJECTION_INFO")) {
        injInputFilename = getenv("INPUT_INJECTION_INFO");
    }
    if (getenv("OUTPUT_INJECTION_LOG")) {
        injOutputFilename = getenv("OUTPUT_INJECTION_LOG");
    }
    if (getenv("INSTRUMENTATION_LIMIT")) {
        limit = std::stoi(getenv("INSTRUMENTATION_LIMIT"));
    }

    // GET_VAR_INT(verbose, "TOOL_VERBOSE", 0, "Enable verbosity inside the tool (1, 2, 3,..)");

    initInstTypeNameMap();

    signal(SIGINT, INThandler); // install Ctrl-C handler

    open_output_file(injOutputFilename);
    if (verbose) printf("nvbit_at_init:end\n");
}

void instrument_function_if_needed(hipCtx_t ctx, hipFunction_t func) {

//    parse_params(injInputFilename);  // injParams are updated based on injection seed file
    parse_flex_grip_file(injInputFilename);
    update_verbose();

    /* Get related functions of the kernel (device function that can be
     * called by the kernel) */
    std::vector<hipFunction_t> related_functions = nvbit_get_related_functions(ctx, func);

    /* add kernel itself to the related function vector */
    related_functions.push_back(func);

    /* iterate on function */
    for (auto f: related_functions) {
        /* "recording" function was instrumented, if set insertion failed
         * we have already encountered this function */
        if (!already_instrumented.insert(f).second) {
            continue;
        }

        std::string kname = removeSpaces(nvbit_get_func_name(ctx, f));
        /* Get the vector of instruction composing the loaded CUFunction "func" */
        const std::vector<Instr *> &instrs = nvbit_get_instrs(ctx, f);

        int maxregs = get_maxregs(f);
        assert(fout.good());
        fout << "Inspecting: " << kname << ";num_static_instrs: " << instrs.size() << ";maxregs: " << maxregs << "("
             << maxregs << ")" << std::endl;
        for (auto i: instrs) {
            std::string opcode = i->getOpcode();
            std::string instTypeStr = extractInstType(opcode);
            int instType = instTypeNameMap[instTypeStr];
            if (verbose) printf("extracted instType: %s, ", instTypeStr.c_str());
            if (verbose) printf("index of instType: %d\n", instTypeNameMap[instTypeStr]);

            /**
             * MODIFICATION FOR FLEXGRIP PF injection
             */
            if ((uint32_t) instType == managed_inj_info_array[0].injInstType ||
                managed_inj_info_array[0].injInstType == NUM_ISA_INSTRUCTIONS) {
                if (verbose) {
                    printf("instruction selected for instrumentation: ");
                    i->print();
                }

                // Tokenize the instruction
                std::vector<std::string> tokens;
                std::string buf; // a buffer string
                std::stringstream ss(i->getSass()); // Insert the string into a stream
                while (ss >> buf)
                    tokens.push_back(buf);

                int destGPRNum = -1;
                int numDestGPRs = 0;

                if (tokens.size() > 1) { // an actual instruction that writes to either a GPR or PR register
                    if (verbose) printf("num tokens = %ld \n", tokens.size());
                    int start = 1; // first token is opcode string
                    if (tokens[0].find('@') != std::string::npos) { // predicated instruction, ignore first token
                        start = 2; // first token is predicate and 2nd token is opcode
                    }

                    // Parse the first operand - this is the first destination
                    int regnum1 = -1;
                    int regtype = extractRegNo(tokens[start], regnum1);
                    if (regtype == 0) { // GPR reg
                        destGPRNum = regnum1;
                        numDestGPRs = (getOpGroupNum(instType) == G_FP64) ? 2 : 1;

                        int szStr = extractSize(opcode);
                        if (szStr == 128) {
                            numDestGPRs = 4;
                        } else if (szStr == 64) {
                            numDestGPRs = 2;
                        }

                        nvbit_insert_call(i, "inject_error", IPOINT_AFTER);
                        nvbit_add_call_arg_const_val64(i, (uint64_t) &managed_inj_info_array[0]);
                        nvbit_add_call_arg_const_val64(i, (uint64_t) &verbose_device);

                        nvbit_add_call_arg_const_val32(i, destGPRNum); // destination GPR register number
                        if (destGPRNum != -1) {
                            nvbit_add_call_arg_reg_val(i, destGPRNum); // destination GPR register val
                        } else {
                            nvbit_add_call_arg_const_val32(i, (unsigned int) -1); // destination GPR register val
                        }
                        nvbit_add_call_arg_const_val32(i, numDestGPRs); // number of destination GPR registers

                        nvbit_add_call_arg_const_val32(i, maxregs); // max regs used by the inst info
                        /**********************************************************************************************/
                    }
                    // If an instruction has two destination registers, not handled!! (TODO: Fix later)
                }
            }
        }
    }
}

/* This call-back is triggered every time a CUDA event is encountered.
 * Here, we identify CUDA kernel launch events and reset the "counter" before
 * th kernel is launched, and print the counter after the kernel has completed
 * (we make sure it has completed by using hipDeviceSynchronize()). To
 * selectively run either the original or instrumented kernel we used
 * nvbit_enable_instrumented() before launching the kernel. */
void nvbit_at_cuda_event(hipCtx_t ctx, int is_exit, nvbit_api_cuda_t cbid,
                         const char *name, void *params, hipError_t *pStatus) {
    /* Identify all the possible CUDA launch events */
    if (cbid == API_CUDA_cuLaunch ||
        cbid == API_CUDA_cuLaunchKernel_ptsz ||
        cbid == API_CUDA_cuLaunchGrid ||
        cbid == API_CUDA_cuLaunchGridAsync ||
        cbid == API_CUDA_cuLaunchKernel) {

        /* cast params to cuLaunch_params since if we are here we know these are
         * the right parameters type */
        auto *p = (cuLaunch_params *) params;

        if (!is_exit) {
            pthread_mutex_lock(&mutex);
            if (kernel_id < limit) {
                instrument_function_if_needed(ctx, p->f);
                // hipDeviceSynchronize();

                nvbit_enable_instrumented(ctx, p->f, true); // run the instrumented version
                // hipDeviceSynchronize();
            } else {
                nvbit_enable_instrumented(ctx, p->f, false); // do not use the instrumented version
            }

        } else {
            if (kernel_id < limit) {
                if (verbose) printf("is_exit\n");
                hipDeviceSynchronize();

                hipError_t le = hipGetLastError();

                std::string kname = removeSpaces(nvbit_get_func_name(ctx, p->f));
                unsigned num_ctas = 0;
                if (cbid == API_CUDA_cuLaunchKernel_ptsz ||
                    cbid == API_CUDA_cuLaunchKernel) {
                    auto *p2 = (cuLaunchKernel_params *) params;
                    num_ctas = p2->gridDimX * p2->gridDimY * p2->gridDimZ;
                }
                assert(fout.good());
                fout << "Injection data; ";
                fout << "index: " << kernel_id << ";";
                fout << "kernel_name: " << kname << ";";
                fout << "ctas: " << num_ctas << ";";
                fout << "selected SM: " << managed_inj_info_array[0].injSMID << ";";
                fout << "selected Lane: " << managed_inj_info_array[0].injLaneID << ";";
                fout << "selected Warp: " << managed_inj_info_array[0].warpID << ";";
                fout << "selected Mask: " << managed_inj_info_array[0].injMask << ";";
                fout << "selected InstType: " << managed_inj_info_array[0].injInstType << ";";
                fout << "injNumActivations: " << managed_inj_info_array[0].injNumActivations << std::endl;

                if (hipSuccess != le) {
                    assert(fout.good());
                    fout << "ERROR FAIL in kernel execution (" << hipGetErrorString(le) << "); " << std::endl;
                    exit(1); // let's exit early
                }

                if (verbose) printf("\n index: %d; kernel_name: %s; \n", kernel_id, kname.c_str());
                kernel_id++; // always increment kernel_id on kernel exit

                // hipDeviceSynchronize();
                pthread_mutex_unlock(&mutex);
            }
        }
    }
}

void nvbit_at_term() {} //nothing to do here
